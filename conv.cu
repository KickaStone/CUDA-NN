#include "conv.cuh"

ConvolutionalLayer::ConvolutionalLayer(int channel_in, int height_in, int width_in, int channel_out, int kernel_size, int dilation, int padding, int stride, hipdnnActivationMode_t activation_mode)
: channel_in(channel_in), height_in(height_in), width_in(width_in), channel_out(channel_out), kernel_size(kernel_size), dilation(dilation), padding(padding), stride(stride), activation_mode(activation_mode)
{
    this->input_size = channel_in * height_in * width_in;

    CUDNN_CHECK(hipdnnCreate(&cudnn_handle));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bias_desc));
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dK_desc));

    int t;
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride, dilation, dilation, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, channel_in, height_in, width_in));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, channel_out, channel_in, kernel_size, kernel_size));
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &t, &channel_out, &height_out, &width_out));

    this->output_size = channel_out * height_out * width_out;

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, channel_out, height_out, width_out));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, channel_out, height_out, height_out));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dK_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, channel_out, channel_in, kernel_size, kernel_size));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act_desc, activation_mode, HIPDNN_PROPAGATE_NAN, 0.0));

    CUDA_CHECK(hipMalloc((void**)&K, sizeof(double) * channel_out * channel_in * kernel_size * kernel_size));
    CUDA_CHECK(hipMalloc((void**)&b, sizeof(double) * channel_out * height_out * width_out));
    CUDA_CHECK(hipMalloc((void**)&a, sizeof(double) * channel_out * height_out * width_out));
    CUDA_CHECK(hipMalloc((void**)&input_grad, sizeof(double) * channel_in * height_in * width_in));
    CUDA_CHECK(hipMalloc((void**)&dK, sizeof(double) * channel_out * channel_in * kernel_size * kernel_size));
    CUDA_CHECK(hipMalloc((void**)&db, sizeof(double) * channel_out * height_out * width_out));

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL);
    hiprandGenerateNormalDouble(generator, K, channel_out * channel_in * kernel_size * kernel_size, 0.0, 1.0);
    hiprandGenerateNormalDouble(generator, b, channel_out * height_out * width_out, 0.0, 1.0);
    
    // CUDA_CHECK(hipMemset(b, 0, sizeof(double) * channel_out * height_out * width_out));
    CUDA_CHECK(hipMemset(a, 0, sizeof(double) * channel_out * height_out * width_out));


    // init backward
    CUDA_CHECK(hipMemset(input_grad, 0, sizeof(double) * channel_in * height_in * width_in));
    CUDA_CHECK(hipMemset(dK, 0, sizeof(double) * channel_out * channel_in * kernel_size * kernel_size));
    CUDA_CHECK(hipMemset(db, 0, sizeof(double) * channel_out * height_out * width_out));

    hiprandDestroyGenerator(generator);
}

ConvolutionalLayer::~ConvolutionalLayer()
{
    CUDNN_CHECK(hipdnnDestroy(cudnn_handle));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bias_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dK_desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act_desc));

    CUDA_CHECK(hipFree(K));
    CUDA_CHECK(hipFree(b));
    CUDA_CHECK(hipFree(a));

    // CUDA_CHECK(hipFree(input_grad));
}

double *ConvolutionalLayer::forward(double *x)
{
    input = x;
    double alpha = 1.0, beta = 0.0;
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn_handle, &alpha, input_desc, input, filter_desc, K, conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, output_desc, a));
    CUDNN_CHECK(hipdnnAddTensor(cudnn_handle, &alpha, bias_desc, b, &alpha, output_desc, a)); 
    CUDNN_CHECK(hipdnnActivationForward(cudnn_handle, act_desc, &alpha, output_desc, a, &beta, output_desc, a));
    return a;
}

double *ConvolutionalLayer::backward(double *output_grad)
{
    double *d_dz;
    CUDA_CHECK(hipMalloc((void**)&d_dz, sizeof(double) * channel_out * height_out * width_out));
    double alpha = 1.0, beta = 0.0;
    CUDNN_CHECK(hipdnnActivationBackward(cudnn_handle, act_desc, &alpha, output_desc, a, output_desc, output_grad, output_desc, a, &beta, output_desc, d_dz));
    
    // for(int i = 0; i < channel_out; i++){
    //     cublasPrintMat(d_dz + i * height_out * width_out, height_out, width_out, "d_dz" + std::to_string(i) + ": ");
    // }
    
    double *db2;
    CUDA_CHECK(hipMalloc((void**)&db2, sizeof(double) * channel_out));
    hipdnnTensorDescriptor_t db2_desc;
    CUDA_CHECK(hipMemset(db2, 0, sizeof(double) * channel_out));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&db2_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(db2_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, channel_out, 1, 1));
    CUDNN_CHECK(hipdnnConvolutionBackwardBias(cudnn_handle, &alpha, output_desc, d_dz, &alpha, db2_desc, db2));

    // CUDNN_CHECK(hipdnnConvolutionBackwardBias(cudnn_handle, &alpha, output_desc, d_dz, &alpha, bias_desc, db));
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(cudnn_handle, &alpha, input_desc, input, output_desc, d_dz, conv_desc, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1, nullptr, 0, &alpha, filter_desc, dK));
    CUDNN_CHECK(hipdnnConvolutionBackwardData(cudnn_handle, &alpha, filter_desc, K, output_desc, d_dz, conv_desc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1, nullptr, 0, &beta, input_desc, input_grad));
    return input_grad;
}

void ConvolutionalLayer::update(double learning_rate)
{
    double alpha = -learning_rate;
    double beta = 1.0;
    CUDNN_CHECK(hipdnnAddTensor(cudnn_handle, &alpha, bias_desc, db, &alpha, bias_desc, b));
    CUDNN_CHECK(hipdnnAddTensor(cudnn_handle, &alpha, dK_desc, dK, &beta, dK_desc, K));
    // reset gradient
    CUDA_CHECK(hipMemset(db, 0, sizeof(double) * channel_out * height_out * width_out));
    CUDA_CHECK(hipMemset(dK, 0, sizeof(double) * channel_out * channel_in * kernel_size * kernel_size));
}
