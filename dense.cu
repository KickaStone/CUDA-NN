#include "dense.cuh"


Dense::Dense(int input_size, int output_size, hipdnnActivationMode_t mode)
{
    CUDA_CHECK(hipMalloc((void **)&weights, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&bias, output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&a, output_size * sizeof(double)));
    this->input_size = input_size;
    this->output_size = output_size;
    this->activation_mode = mode;

    hiprandGenerator_t curand_generator;
    hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_generator, HIPRAND_RNG_PSEUDO_MT19937);
    hiprandGenerateNormalDouble(curand_generator, weights, input_size * output_size, 0, 0.1);
    hiprandGenerateNormalDouble(curand_generator, bias, output_size, 0, 0.1);

    CUDA_CHECK(hipMalloc((void **)&d_weights, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&d_bias, output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&input_grad, input_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_weights, 0.0, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_bias, 0.0, output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(input_grad, 0.0, input_size * sizeof(double)));

    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    CUDNN_CHECK(hipdnnCreate(&cudnn_handle));

    hiprandDestroyGenerator(curand_generator);

    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act_desc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act_desc, activation_mode, HIPDNN_PROPAGATE_NAN, 0));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, input_size, 1));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, output_size, 1));
}

Dense::~Dense()
{
    try{
        CUDA_CHECK(hipFree(weights));
        CUDA_CHECK(hipFree(bias));
        CUDA_CHECK(hipFree(a));
        CUDA_CHECK(hipFree(d_weights));
        CUDA_CHECK(hipFree(d_bias));
        CUDA_CHECK(hipFree(input_grad));

        CUBLAS_CHECK(hipblasDestroy(cublas_handle));
        CUDNN_CHECK(hipdnnDestroy(cudnn_handle));
        CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act_desc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    }catch(const std::exception& e){
        std::cerr << e.what() << '\n';
    }
}

double* Dense::forward(double *input_data) {
    this->input = input_data;
    double alpha = 1.0;
    double beta = 0.0;
    CUDA_CHECK(hipMemset(a, 0.0, output_size * sizeof(double)));

    // a = weights * input
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             output_size, 1, input_size,
                             &alpha, weights, input_size, input_data, input_size,
                             &beta, a, output_size));
//    cublasPrintMat(bias, output_size, 1);
//    cublasPrintMat(a, output_size, 1);

    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, bias, 1, a, 1));
    CUDNN_CHECK(hipdnnActivationForward(cudnn_handle, act_desc, &alpha, output_desc, a, &beta, output_desc, a));
    return a;
}

double* Dense::backward(double *output_grad) {


    double *d_dz;
    CUDA_CHECK(hipMalloc((void **)&d_dz, output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_dz, 0.0, output_size * sizeof(double)));

    double alpha = 1.0;
    double beta = 0.0;
    CUDNN_CHECK(hipdnnActivationBackward(cudnn_handle, act_desc, &alpha, output_desc, a, output_desc, output_grad, input_desc, input, &beta, output_desc, d_dz));
    
    // std::cout << "d_dz" << std::endl;
    // cublasPrintMat(d_dz, output_size, 1);
    
    // calculate d_weights
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             input_size, output_size, 1,
                             &alpha, input, input_size, d_dz, 1,
                             &alpha, d_weights, input_size));

    // std::cout << "d_weights" << std::endl;
    // cublasPrintMat(d_weights, input_size, output_size);

    // calculate d_bias
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, d_dz, 1, d_bias, 1));
    // cublasPrintMat(d_bias, output_size, 1, "d_bias");

    // calculate d_input
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             input_size, 1, output_size,
                             &alpha, weights, input_size, d_dz, output_size,
                             &beta, input_grad, input_size));
    // std::cout << "d_input" << std::endl;
    // cublasPrintMat(input_grad, input_size, 1);
    CUDA_CHECK(hipFree(d_dz));
    return input_grad;
}

void Dense::update(double lr) {
    // update weights and bias
    double alpha = -lr;
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, input_size * output_size, &alpha, d_weights, 1, weights, 1));
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, d_bias, 1, bias, 1));

    // reset d_weights and d_bias
    CUDA_CHECK(hipMemset(d_weights, 0.0, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_bias, 0.0, output_size * sizeof(double)));
}




