#include "dense.cuh"


Dense::Dense(int input_size, int output_size, hipdnnActivationMode_t mode)
{
    CUDA_CHECK(hipMalloc((void **)&weights, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&bias, output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&a, output_size * sizeof(double)));
    this->input_size = input_size;
    this->output_size = output_size;
    this->activation_mode = mode;

    hiprandGenerator_t curand_generator;
    hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_generator, HIPRAND_RNG_PSEUDO_MT19937);
    hiprandGenerateNormalDouble(curand_generator, weights, input_size * output_size, 0, 0.1);
    hiprandGenerateNormalDouble(curand_generator, bias, output_size, 0, 0.1);

    CUDA_CHECK(hipMalloc((void **)&d_weights, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&d_bias, output_size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&input_grad, input_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_weights, 0.0, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_bias, 0.0, output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(input_grad, 0.0, input_size * sizeof(double)));

    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    CUDNN_CHECK(hipdnnCreate(&cudnn_handle));

    hiprandDestroyGenerator(curand_generator);
}

Dense::~Dense()
{
    try{
        CUDA_CHECK(hipFree(weights));
        CUDA_CHECK(hipFree(bias));
        CUDA_CHECK(hipFree(a));
        CUDA_CHECK(hipFree(d_weights));
        CUDA_CHECK(hipFree(d_bias));
        CUDA_CHECK(hipFree(input_grad));

        CUBLAS_CHECK(hipblasDestroy(cublas_handle));
        CUDNN_CHECK(hipdnnDestroy(cudnn_handle));
    }catch(const std::exception& e){
        std::cerr << e.what() << '\n';
    }
}

double* Dense::forward(double *input_data) {
    this->input = input_data;
    double alpha = 1.0;
    double beta = 0.0;
    CUDA_CHECK(hipMemset(a, 0.0, output_size * sizeof(double)));

    // a = weights * input
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             output_size, 1, input_size,
                             &alpha, weights, input_size, input_data, input_size,
                             &beta, a, output_size));
//    cublasPrintMat(bias, output_size, 1);
//    cublasPrintMat(a, output_size, 1);


    // a += bias
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, bias, 1, a, 1));

//    cublasPrintMat(a, output_size, 1, "a");

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, output_size, 1));

    hipdnnActivationDescriptor_t act_desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act_desc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0));
    CUDNN_CHECK(hipdnnActivationForward(cudnn_handle, act_desc, &alpha, desc, a, &beta, desc, a));

    hipdnnDestroyTensorDescriptor(desc);
    hipdnnDestroyActivationDescriptor(act_desc);
    return a;
}

double* Dense::backward(double *output_grad) {

    hipdnnActivationDescriptor_t act_desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act_desc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0));
    
    hipdnnTensorDescriptor_t a_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&a_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(a_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, output_size, 1));

    hipdnnTensorDescriptor_t d_a_desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&d_a_desc));

    double *d_dz;
    CUDA_CHECK(hipMalloc((void **)&d_dz, output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_dz, 0.0, output_size * sizeof(double)));

    double alpha = 1.0;
    double beta = 0.0;
    CUDNN_CHECK(hipdnnActivationBackward(cudnn_handle, act_desc, &alpha, a_desc, a, a_desc, output_grad, a_desc, input, &beta, a_desc, d_dz));
    
    // std::cout << "d_dz" << std::endl;
    // cublasPrintMat(d_dz, output_size, 1);
    
    // calculate d_weights
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             input_size, output_size, 1,
                             &alpha, input, input_size, d_dz, 1,
                             &alpha, d_weights, input_size));

    // std::cout << "d_weights" << std::endl;
    // cublasPrintMat(d_weights, input_size, output_size);

    // calculate d_bias
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, d_dz, 1, d_bias, 1));
    // cublasPrintMat(d_bias, output_size, 1, "d_bias");


    // calculate d_input
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             input_size, 1, output_size,
                             &alpha, weights, input_size, d_dz, output_size,
                             &beta, input_grad, input_size));
    // std::cout << "d_input" << std::endl;
    // cublasPrintMat(input_grad, input_size, 1);

    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act_desc));
    return input_grad;
}

void Dense::update(double lr) {
    // update weights and bias
    double alpha = -lr;
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, input_size * output_size, &alpha, d_weights, 1, weights, 1));
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, output_size, &alpha, d_bias, 1, bias, 1));

    // reset d_weights and d_bias
    CUDA_CHECK(hipMemset(d_weights, 0.0, input_size * output_size * sizeof(double)));
    CUDA_CHECK(hipMemset(d_bias, 0.0, output_size * sizeof(double)));
}




