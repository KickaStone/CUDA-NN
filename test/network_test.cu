#include <gtest/gtest.h>
#include "../mnist_loader.h"
#include "../network.cuh"
#include "../dense.cuh"
#include "../conv.cuh"
#include "../maxpooling.cuh"


std::vector<double*> train_data, test_data;
std::vector<int> train_label, test_label;
TEST(Network, load_mnist) {
    const char* train_image_file = "E:/Projects/Cuda/CUDA-NN/data/train-images.idx3-ubyte";
    const char* train_label_file = "E:/Projects/Cuda/CUDA-NN/data/train-labels.idx1-ubyte";
    const char* test_image_file = "E:/Projects/Cuda/CUDA-NN/data/t10k-images.idx3-ubyte";
    const char* test_label_file = "E:/Projects/Cuda/CUDA-NN/data/t10k-labels.idx1-ubyte";

    load_mnist(train_image_file, train_label_file, train_data, train_label);
    load_mnist(test_image_file, test_label_file, test_data, test_label);

    ASSERT_EQ(train_data.size(), 60000);
    ASSERT_EQ(train_label.size(), 60000);
    ASSERT_EQ(test_data.size(), 10000);
    ASSERT_EQ(test_label.size(), 10000);
}

TEST(Network, cnn){
    NeuralNetwork nn = NeuralNetwork(7, 784, 10);
    nn.add_layer(new ConvolutionalLayer(1, 28, 28, 6, 5, 1, 2, 1, HIPDNN_ACTIVATION_RELU));
    nn.add_layer(new MaxPooling(6, 28, 28, 2, 2, 0, 1));
    nn.add_layer(new ConvolutionalLayer(6, 14, 14, 16, 5, 1, 0, 1, HIPDNN_ACTIVATION_RELU));
    nn.add_layer(new MaxPooling(16, 10, 10, 2, 2, 0, 1));
    nn.add_layer(new Dense(5 * 5 * 16, 120, HIPDNN_ACTIVATION_SIGMOID));
    nn.add_layer(new Dense(120, 84, HIPDNN_ACTIVATION_SIGMOID));
    nn.add_layer(new Dense(84, 10, HIPDNN_ACTIVATION_SIGMOID));
    nn.setData(train_data, train_label, test_data, test_label);
    nn.setParams(30, 10, 0.1, 0);
    nn.train();
}