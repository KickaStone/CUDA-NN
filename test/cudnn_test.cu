#include <gtest/gtest.h>
#include <hipDNN.h>
#include <Eigen/Dense>
#include <iostream>
#include <hipblas.h>


using namespace Eigen;
using namespace std;

TEST(cudnn, actBack){

    VectorXd a(3);
    a << 0.1, 0.2, 0.3;

    cout << a.unaryExpr([](double x){return x * (1-x);}) << endl;
    double *d_a;
    

    hipMalloc((void**)&d_a, 3 * sizeof(double));

    hipdnnHandle_t handle;
    hipdnnCreate(&handle);
    hipdnnTensorDescriptor_t desc;
    hipdnnCreateTensorDescriptor(&desc);
    hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, 1, 3);
    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasSetMatrix(8, 1, sizeof(double), a.data(), 8, d_a, 8);


    hipdnnActivationBackward(handle, actDesc, &alpha, desc, d_a, desc, d_a, desc, d_a, &beta, desc, d_a);
    hipDeviceSynchronize();
    hipFree(d_a);
    hipblasGetMatrix(8, 1, sizeof(double), d_a, 8, a.data(), 8);
    cout << a << endl;
    
    hipdnnDestroyTensorDescriptor(desc);

}