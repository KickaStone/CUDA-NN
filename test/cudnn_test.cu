#include <gtest/gtest.h>
#include <hipDNN.h>
#include <Eigen/Dense>
#include <iostream>
#include <hipblas.h>

#include "../common.h"


using namespace Eigen;
using namespace std;

TEST(cudnn, actBack){

    VectorXd a(3);
    a << 0.1, 0.2, 0.3;

    cout << a.unaryExpr([](double x){return x * (1-x);}) << endl;
    double *d_a;
    

    hipMalloc((void**)&d_a, 3 * sizeof(double));

    hipdnnHandle_t handle;
    hipdnnCreate(&handle);
    hipdnnTensorDescriptor_t desc;
    hipdnnCreateTensorDescriptor(&desc);
    hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, 1, 3);
    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasSetMatrix(8, 1, sizeof(double), a.data(), 8, d_a, 8);


    hipdnnActivationBackward(handle, actDesc, &alpha, desc, d_a, desc, d_a, desc, d_a, &beta, desc, d_a);
    hipDeviceSynchronize();
    hipFree(d_a);
    hipblasGetMatrix(8, 1, sizeof(double), d_a, 8, a.data(), 8);
    cout << a << endl;
    
    hipdnnDestroyTensorDescriptor(desc);

}

TEST(cudnn, conv){
    
        MatrixXd a(3, 3);
        a << 1, 2, 3,
            4, 5, 6,
            7, 8, 9;

        MatrixXd b(2, 2);
        b << 1, 2,
            3, 4;

        double *d_a;
        double *d_b;
        double *d_c;

        CUDA_CHECK(hipMalloc((void**)&d_a, 9 * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&d_b, 4 * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&d_c, 4 * sizeof(double)));

        CUBLAS_CHECK(hipblasSetMatrix(9, 1, sizeof(double), a.data(), 9, d_a, 9));
        CUBLAS_CHECK(hipblasSetMatrix(4, 1, sizeof(double), b.data(), 4, d_b, 4));

        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        hipdnnTensorDescriptor_t descA;
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&descA));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(descA, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, 1, 3, 3));

        hipdnnFilterDescriptor_t descB;
        CUDNN_CHECK(hipdnnCreateFilterDescriptor(&descB));
        CUDNN_CHECK(hipdnnSetFilter4dDescriptor(descB, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, 1, 1, 2, 2));

        hipdnnConvolutionDescriptor_t descConv;
        CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&descConv));
        CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(descConv, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE));


        
}